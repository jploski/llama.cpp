#include "hip/hip_runtime.h"
#include "ssm_scan.cuh"

template <int block_size>
static __global__ void ssm_scan_f32(
    const float * src0, const float * src1, const float * src2, const float * src3,
    const float * src4, const float * src5, const float * src6,
    const int src0_nb1, const int src0_nb2,
    const int src1_nb0, const int src1_nb1, const int src1_nb2,
    const int src2_nb0, const int src2_nb1,
    const int src3_nb1,
    const int src4_nb1,
    const int src5_nb1,
    const int src6_nb1,
    float * dst,
    const int nc, const int nr, const int n_t, const int n_kv) {

//    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    const int ith = tid;
    const int nth = WARP_SIZE;

    // rows per thread
    const int dr = (nr + nth - 1)/nth;

    // row range for this thread
    const int ir0 = dr*ith;
    const int ir1 = min(ir0 + dr, nr);
    const int ir  = ir1 - ir0;

    if (n_kv > 1) {
        // it's hard to know if the source states have already been copied
        // when there are multiple, so copy them already.
        for (int i3 = 0; i3 < n_kv; ++i3) {
            float * s0 = (float *) ((char *) src0 + ir0*src0_nb1 + i3*src0_nb2);
            float * s  = (float *) ((char *)  dst + ir0*src0_nb1 + i3*src0_nb2 + src1_nb2);

            //memcpy(s, s0, nc*ir*sizeof(float));
            for (int i4 = 0; i4 < nc*ir; i4++) {
                s[i4] = s0[i4];
            }
        }
    }

    for (int i2 = 0; i2 < n_t; ++i2) {
        int32_t * sq = (int32_t *) ((char *) src6 +  i2*src6_nb1); // {n_kv, n_tokens}
        float *   y  = (float *)   ((char *)  dst + ir0*src1_nb0 +    i2*src1_nb1); // {d_inner, n_tokens}
        float *   s  = (float *)   ((char *)  dst + ir0*src0_nb1 + sq[0]*src0_nb2 + src1_nb2); // {d_state, d_inner, n_kv}
        float *   s0;
        float *   x  = (float *)   ((char *) src1 + ir0*src1_nb0 + i2*src1_nb1); // {d_inner, n_tokens}
        float *   dt = (float *)   ((char *) src2 + ir0*src2_nb0 + i2*src2_nb1); // {d_inner, n_tokens}
        float *   A  = (float *)   ((char *) src3 + ir0*src3_nb1); // {d_state, d_inner}
        float *   B  = (float *)   ((char *) src4 +  i2*src4_nb1); // {d_state, n_tokens}
        float *   C  = (float *)   ((char *) src5 +  i2*src5_nb1); // {d_state, n_tokens}

        // avoid needing to copy the state for the first token
        if (i2 == 0) {
            s0 = (float *) ((char *) src0 + ir0*(src0_nb1) + sq[0]*src0_nb2); // {d_state, d_inner, n_kv}
        } else {
            // otherwise the source is the same as the destination
            s0 = s;
        }

        // d_inner
        for (int i1 = 0; i1 < ir; ++i1) {
            // ref: https://github.com/state-spaces/mamba/blob/34076d664838588a3c97727b263478ab9f621a07/mamba_ssm/ops/triton/selective_state_update.py#L78
            float dt_soft_plus = dt[i1] <= 20.0f ? log1pf(expf(dt[i1])) : dt[i1];
            float x_dt = x[i1] * dt_soft_plus;
            float sumf = 0.0f;
            // d_state
            for (int i0 = 0; i0 < nc; ++i0) {
                int i = i0 + i1*nc;
                // state = prev_state * dA + dB * x
                float state = (s0[i] * expf(dt_soft_plus * A[i])) + (B[i0] * x_dt);
                // y = rowwise_dotprod(state, C)
                sumf += state * C[i0];
                s[i] = state;
            }
            y[i1] = sumf;
        }

        // handle copies when there are multiple output states
        for (int i3 = 1; i3 < n_kv; ++i3) {
            int32_t seq = sq[i3];
            if (0 <= seq && seq < n_kv) {
                float * s1 = s + (seq - sq[0])*nc*nr;
                //memcpy(s1, s, nc*ir*sizeof(float));
                for (int i4 = 0; i4 < nc*ir; i4++) {
                    s1[i4] = s[i4];
                }
            } else {
                // stop at negative or too big seq_ids
                break;
            }
        }
    }
}

static void ssm_scan_f32_cuda(
    const float * src0, const float * src1, const float * src2, const float * src3,
    const float * src4, const float * src5, const float * src6,
    const int src0_nb1, const int src0_nb2,
    const int src1_nb0, const int src1_nb1, const int src1_nb2,
    const int src2_nb0, const int src2_nb1,
    const int src3_nb1,
    const int src4_nb1,
    const int src5_nb1,
    const int src6_nb1,
    float * dst,
    const int nc, const int nr, const int n_t, const int n_kv, hipStream_t stream) {

    const dim3 block_dims(WARP_SIZE, 1, 1);
    const int nblocks = 1; // TODO

    ssm_scan_f32<WARP_SIZE><<<nblocks, block_dims, 0, stream>>>(
        src0, src1, src2, src3, src4, src5, src6,
        src0_nb1, src0_nb2,
        src1_nb0, src1_nb1, src1_nb2,
        src2_nb0, src2_nb1,
        src3_nb1,
        src4_nb1,
        src5_nb1,
        src6_nb1,
        dst,
        nc, nr, n_t, n_kv);
}

void ggml_cuda_op_ssm_scan(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const struct ggml_tensor * src0 = dst->src[0]; // s
    const struct ggml_tensor * src1 = dst->src[1]; // x
    const struct ggml_tensor * src2 = dst->src[2]; // dt
    const struct ggml_tensor * src3 = dst->src[3]; // A
    const struct ggml_tensor * src4 = dst->src[4]; // B
    const struct ggml_tensor * src5 = dst->src[5]; // C
    const struct ggml_tensor * src6 = dst->src[6]; // sq

    const int64_t nc   = src0->ne[0]; // d_state
    const int64_t nr   = src0->ne[1]; // d_inner
    const int64_t n_t  = src1->ne[1]; // number of tokens in the batch
    const int64_t n_kv = src0->ne[2]; // max number of sequences in the batch

    GGML_ASSERT(ggml_nelements(src1) + ggml_nelements(src0) == ggml_nelements(dst));
    GGML_ASSERT(src0->nb[0] == sizeof(float));
    GGML_ASSERT(src1->nb[0] == sizeof(float));
    GGML_ASSERT(src2->nb[0] == sizeof(float));
    GGML_ASSERT(src3->nb[0] == sizeof(float));
    GGML_ASSERT(src4->nb[0] == sizeof(float));
    GGML_ASSERT(src5->nb[0] == sizeof(float));
    // required for the dot product between s and C, and when copying the states
    GGML_ASSERT(src0->nb[1] == src0->ne[0]*sizeof(float));
    // required for per-sequence offsets for states
    GGML_ASSERT(src0->nb[2] == src0->ne[0]*src0->ne[1]*sizeof(float));
    // required to get correct offset for state destination (i.e. src1->nb[2])
    GGML_ASSERT(src1->nb[2] == src1->ne[0]*src1->ne[1]*sizeof(float));

    const float * src0_d = (const float *)src0->data;
    const float * src1_d = (const float *)src1->data;
    const float * src2_d = (const float *)src2->data;
    const float * src3_d = (const float *)src3->data;
    const float * src4_d = (const float *)src4->data;
    const float * src5_d = (const float *)src5->data;
    const float * src6_d = (const float *)src6->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    ssm_scan_f32_cuda(
        src0_d, src1_d, src2_d, src3_d, src4_d, src5_d, src6_d,
        src0->nb[1], src0->nb[2],
        src1->nb[0], src1->nb[1], src1->nb[2],
        src2->nb[0], src2->nb[1],
        src3->nb[1],
        src4->nb[1],
        src5->nb[1],
        src6->nb[1],
        dst_d,
        nc, nr, n_t, n_kv, stream);
}
