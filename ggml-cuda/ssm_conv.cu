#include "hip/hip_runtime.h"
#include "ssm_conv.cuh"

template <int block_size>
static __global__ void ssm_conv_f32(
    const float * src0, const float * src1, const float * src2, const float * src3,
    const int src0_ne0, const int src0_nb1, const int src0_nb2,
    const int src1_nb0, const int src1_nb1,
    const int src2_nb1, const int src2_nb2,
    const int src3_nb1,
    float * dst,
    const int nc, const int nr, const int n_t, const int n_kv) {

//    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    const int ith = tid;
    const int nth = WARP_SIZE;

    // rows per thread
    const int dr = (nr + nth - 1)/nth;

    // row range for this thread
    const int ir0 = dr*ith;
    const int ir1 = min(ir0 + dr, nr);
    const int ir  = ir1 - ir0;

    if (n_kv > 1) {
        // multiple sequences means it's hard to know when it's the first time a state is read,
        // so copy them all over to the destination, just to be sure.
        for (int i3 = 0; i3 < n_kv; ++i3) {
            float * s0 = (float *) ((char *) src0 + ir0*src0_nb1 + i3*src0_nb2);
            float * s  = (float *) ((char *)  dst + ir0*src2_nb1 + i3*src2_nb2 + nr*n_t*sizeof(float));
            // can't use memcpy because of d_conv vs d_conv - 1
            for (int i1 = 0; i1 < ir; ++i1) {
                for (int i0 = 0; i0 < nc - 1; ++i0) {
                    // copy s0 to last (d_conv - 1) columns of s
                    s[1 + i0 + i1*nc] = s0[i0 + i1*(nc - 1)];
                }
            }
        }
    }

    for (int i2 = 0; i2 < n_t; ++i2) {
        int32_t * sq = (int32_t *) ((char *) src3 +  i2*src3_nb1); // {n_kv, n_tokens}
        float *   x  = (float *)   ((char *)  dst + ir0*sizeof(float) + i2*(nr*sizeof(float))); // {d_inner, n_tokens}
        float *   s  = (float *)   ((char *)  dst + ir0*src2_nb1 + sq[0]*src2_nb2 + nr*n_t*sizeof(float)); // {d_conv, d_inner, n_kv}
        float *   s0; // {d_conv - 1, d_inner, n_kv}
        float *   x0 = (float *)   ((char *) src1 + ir0*src1_nb0 + i2*src1_nb1); // {d_inner, n_tokens}
        float *   c  = (float *)   ((char *) src2 + ir0*src2_nb1); // {d_conv, d_inner}
        int ne0s0;

        // avoid needing to copy the state for the first token
        if (i2 == 0) {
            s0 = (float *) ((char *) src0 + ir0*src0_nb1 + sq[0]*src0_nb2); // {d_conv - 1, d_inner, n_kv}
            ne0s0 = src0_ne0;
        } else {
            // the source is the last (d_conv - 1) columns of the destination
            s0 = s + 1;
            ne0s0 = nc;
        }

        // d_inner
        for (int i1 = 0; i1 < ir; ++i1) {
            // shift state left
            for (int i0 = 0; i0 < nc - 1; ++i0) {
                s[i0 + i1*nc] = s0[i0 + i1*ne0s0];
            }
            // insert x on the last column
            s[(nc - 1) + i1*nc] = x0[i1];
        }

        // handle copies when there are multiple output states
        for (int i3 = 1; i3 < n_kv; ++i3) {
            int32_t seq = sq[i3];
            if (0 <= seq && seq < n_kv) {
                float * s1 = s + (seq - sq[0])*nc*nr;

                //memcpy(s1, s, nc*ir*sizeof(float));
                for (int i4 = 0; i4 < nc*ir; i4++) {
                    s1[i4] = s[i4];
                }
            } else {
                // stop at negative or too big seq_ids
                break;
            }
        }

        // it seems a little faster when this is separate from the state shift
        for (int i1 = 0; i1 < ir; ++i1) {
            // rowwise dot product
            float sumf = 0.0f;
            for (int i0 = 0; i0 < nc; ++i0) {
                int i = i0 + i1*nc;
                sumf += s[i] * c[i];
            }
            x[i1] = sumf;
        }
    }
}

static void ssm_conv_f32_cuda(
    const float * src0, const float * src1, const float * src2, const float * src3,
    const int src0_ne0, const int src0_nb1, const int src0_nb2,
    const int src1_nb0, const int src1_nb1,
    const int src2_nb1, const int src2_nb2,
    const int src3_nb1,
    float * dst,
    const int nc, const int nr, const int n_t, const int n_kv, hipStream_t stream) {

    const dim3 block_dims(WARP_SIZE, 1, 1);
    const int nblocks = 1; // TODO

    ssm_conv_f32<WARP_SIZE><<<nblocks, block_dims, 0, stream>>>(
        src0, src1, src2, src3,
        src0_ne0, src0_nb1, src0_nb2,
        src1_nb0, src1_nb1,
        src2_nb1, src2_nb2,
        src3_nb1,
        dst,
        nc, nr, n_t, n_kv);
}

void ggml_cuda_op_ssm_conv(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const struct ggml_tensor * src0 = dst->src[0]; // conv_state
    const struct ggml_tensor * src1 = dst->src[1]; // x
    const struct ggml_tensor * src2 = dst->src[2]; // conv1d.weight
    const struct ggml_tensor * src3 = dst->src[3]; // state_seq

    const int nc   = src2->ne[0]; // d_conv
    const int nr   = src0->ne[1]; // d_inner
    const int n_t  = src1->ne[1]; // n_tokens
    const int n_kv = src0->ne[2]; // max number of sequences in the batch

    GGML_ASSERT((nr*n_t) + (nc*nr*n_kv) == ggml_nelements(dst));
    GGML_ASSERT(src0->nb[0] == sizeof(float));
    GGML_ASSERT(src1->nb[0] == sizeof(float));
    GGML_ASSERT(src2->nb[0] == sizeof(float));
    GGML_ASSERT(src3->nb[0] == sizeof(int32_t));
    GGML_ASSERT(src0->nb[1] == src0->ne[0]*sizeof(float));
    // for use with the destination state offset between sequences
    GGML_ASSERT(src2->nb[2] == src2->ne[1]*src2->ne[0]*sizeof(float));

    const float * src0_d = (const float *)src0->data;
    const float * src1_d = (const float *)src1->data;
    const float * src2_d = (const float *)src2->data;
    const float * src3_d = (const float *)src3->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    ssm_conv_f32_cuda(src0_d, src1_d, src2_d, src3_d,
        src0->ne[0], src0->nb[1], src0->nb[2],
        src1->nb[0], src1->nb[1],
        src2->nb[1], src2->nb[2],
        src3->nb[1],
        dst_d, nc, nr, n_t, n_kv, stream);
}
